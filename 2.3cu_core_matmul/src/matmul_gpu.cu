#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
#include "matmul.hpp"

__global__ void MatmulKernel(float *M_device, float *N_device, float *P_device, int width){

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float P_element = 0;

    for (int k = 0; k < width; k ++){
        float M_element = M_device[y * width + k];
        float N_element = N_device[k * width + x];
        P_element += M_element * N_element;
    }

    P_device[y * width + x] = P_element;
}

void MatmulOnDevice(float *M_host, float *N_host, float* P_host, int width, int blockSize){
    int size = width * width * sizeof(float);

    float *M_device;
    float *N_device;

    hipMalloc(&M_device, size);
    hipMalloc(&N_device, size);

    hipMemcpy(M_device, M_host, size, hipMemcpyHostToDevice);
    hipMemcpy(N_device, N_host, size, hipMemcpyHostToDevice);

    float *P_device;
    hipMalloc(&P_device, size);

    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid(width / blockSize, width / blockSize);
    MatmulKernel <<<dimGrid, dimBlock>>> (M_device, N_device, P_device, width);

    hipMemcpy(P_host, P_device, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    /* Free */
    hipFree(P_device);
    hipFree(N_device);
    hipFree(M_device);
}

