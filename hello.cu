#include <iostream>
#include <hip/hip_runtime.h>

// CUDA 核函数：每个线程执行一次加法
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int N = 512;
    size_t size = N * sizeof(float);

    // 分配主机内存
    float *h_a = (float*)malloc(size);
    float *h_b = (float*)malloc(size);
    float *h_c = (float*)malloc(size);

    // 初始化主机数据
    for (int i = 0; i < N; ++i) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    // 分配设备内存
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // 将数据从主机复制到设备
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // 启动 CUDA 核函数
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // 等待 GPU 完成
    hipDeviceSynchronize();

    // 将结果从设备复制回主机
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // 验证结果
    for (int i = 0; i < 5; ++i) {
        std::cout << "h_c[" << i << "] = " << h_c[i] << std::endl;
    }

    // 释放内存
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
